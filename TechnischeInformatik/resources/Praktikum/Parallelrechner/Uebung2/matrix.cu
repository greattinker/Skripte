#include "hip/hip_runtime.h"
//#include "stdafx.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "errorCheck.h"

// Kernel that executes on the CUDA device
__global__ void addElems(int *a, int *b, int *c, int N)
{
  int idx = threadIdx.x;
  if (idx<N) c[idx] = a[idx] + b[idx];
}

// main routine that executes on the host
int main(void)
{
  int *a_device, *a_host;  
  int *b_device, *b_host; 
  int *c_device, *c_host;  
  
  const int N = 1000000;  // Number of elements in arrays
  size_t size = N * sizeof(int);
  
  a_host = (int *)malloc(size);  
  b_host = (int *)malloc(size); 
  c_host = (int *)malloc(size); 
  
  // Allocate array on host
  CudaSafeCall(hipMalloc((void **) &a_device, size)); CudaCheckError();   // Allocate array on device
  CudaSafeCall(hipMalloc((void **) &b_device, size)); CudaCheckError();   // Allocate array on device
  CudaSafeCall(hipMalloc((void **) &c_device, size)); CudaCheckError();   // Allocate array on device
  
  // Initialize host array and copy it to CUDA device
  for (int i=0; i<N; i++){ a_host[i] = (int)i; b_host[i] = (int)i;}
  
  CudaSafeCall(hipMemcpy(a_device, a_host, size, hipMemcpyHostToDevice)); CudaCheckError();
  CudaSafeCall(hipMemcpy(b_device, b_host, size, hipMemcpyHostToDevice)); CudaCheckError();
 
  // Do calculation on device:
  int blockX = N;
  int blockY = 1;
  int blockZ = 1;
  int block_size = blockX*blockY*blockZ;
  int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
  dim3 block(blockX, blockY, blockZ); 
  
  addElems <<< n_blocks, block >>> (a_device, b_device, c_device, N);
  
  
  // Retrieve result from device and store it in host array
  CudaSafeCall(hipMemcpy(c_host, c_device, size, hipMemcpyDeviceToHost));
  CudaCheckError();
  // Print results
  for (int i=0; i<N; i++) printf("%d %d %d %d\n", i, a_host[i], b_host[i], c_host[i]);
  
  
  // Cleanup
  free(a_host); CudaSafeCall(hipFree(a_device)); CudaCheckError();
  free(b_host); CudaSafeCall(hipFree(b_device)); CudaCheckError();
  free(c_host); CudaSafeCall(hipFree(c_device)); CudaCheckError();
}
