#include "hip/hip_runtime.h"
//#include "stdafx.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "errorCheck.h"


// Kernel that executes on the CUDA device
__global__ void addElems(int *a, int *b, int *c, int N)
{
//  int idx = (threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y) + (blockDim.x*blockDim.y*blockDim.z) * ( blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y);
	int idx = threadIdx.x + blockDim.x * blockIdx.x ;
  	if (idx<N) c[idx] = a[idx] + b[idx];
}

// main routine that executes on the host
int main(void)
{
	int *a_device, *a_host;  
	int *b_device, *b_host; 
	int *c_device, *c_host;  

	const int N = 1000000;  // Number of elements in arrays
	size_t size = N * sizeof(int);

	// Allocate arrays on host
	a_host = (int *)malloc(size);  
	b_host = (int *)malloc(size); 
	c_host = (int *)malloc(size); 

	// Allocate arrays on device
	CudaSafeCall(hipMalloc((void **) &a_device, size)); CudaCheckError();   
	CudaSafeCall(hipMalloc((void **) &b_device, size)); CudaCheckError();  
	CudaSafeCall(hipMalloc((void **) &c_device, size)); CudaCheckError();  

	// Initialize host arrays and copy it to CUDA device
	for (int i=0; i<N; i++){ 
		a_host[i] = (int)i; 
		b_host[i] = (int)i;
	}
  
	CudaSafeCall(hipMemcpy(a_device, a_host, size, hipMemcpyHostToDevice)); CudaCheckError();
	CudaSafeCall(hipMemcpy(b_device, b_host, size, hipMemcpyHostToDevice)); CudaCheckError();

	// Do calculation on device:
	dim3 grid(16, 1, 1); 
	dim3 block(65500,  1,  1); 
  
  	addElems <<< grid, block >>> (a_device, b_device, c_device, N);
  
  
  	// Retrieve result from device and store it in host array
 	 CudaSafeCall(hipMemcpy(c_host, c_device, size, hipMemcpyDeviceToHost));
	//CudaCheckError();

	// Print results
	for (int i=0; i<N; i++) printf("%d %d %d %d\n", i, a_host[i], b_host[i], c_host[i]);


	// Cleanup
	free(a_host); CudaSafeCall(hipFree(a_device)); CudaCheckError();
	free(b_host); CudaSafeCall(hipFree(b_device)); CudaCheckError();
	free(c_host); CudaSafeCall(hipFree(c_device)); CudaCheckError();
}
