//#include "stdafx.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "errorCheck.h"



// main routine that executes on the host
int main(void)
{
	int *a_device, *a_host;  
	int *b_device, *b_host;  

	const int N = 1000000;  // Number of elements in arrays
	size_t size = N * sizeof(int);

	// Allocate arrays on host
	a_host = (int *)malloc(size);  
	b_host = (int *)malloc(size); 

	// Allocate arrays on device
	CudaSafeCall(hipMalloc((void **) &a_device, size)); CudaCheckError();   
	CudaSafeCall(hipMalloc((void **) &b_device, size)); CudaCheckError();   

	// Initialize host arrays and copy it to CUDA device
	for (int i=0; i<N; i++){ 
		a_host[i] = (int)i; 
	}
  
	CudaSafeCall(hipMemcpy(a_device, a_host, size, hipMemcpyHostToDevice)); CudaCheckError();
	CudaSafeCall(hipMemcpy(b_device, a_device, size, hipMemcpyDeviceToDevice)); CudaCheckError();


  	// Retrieve result from device and store it in host array
 	CudaSafeCall(hipMemcpy(b_host, b_device, size, hipMemcpyDeviceToHost));
	CudaCheckError();

	// Print results
	for (int i=0; i<N; i++) printf("%d %d %d\n", i, a_host[i], b_host[i]);


	// Cleanup
	free(a_host); CudaSafeCall(hipFree(a_device)); CudaCheckError();
	free(b_host); CudaSafeCall(hipFree(b_device)); CudaCheckError();
}
