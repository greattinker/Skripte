#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <time.h>
#include <sys/time.h>
#include "errorCheck.h"

#define GRIDDIMX 32
#define GRIDDIMY 32
#define GRIDDIMZ 1

#define BLOCKDIMX 32
#define BLOCKDIMY 32
#define BLOCKDIMZ 1


static inline uint32_t* random_mat( uint32_t n );
static inline uint32_t* zero_mat( uint32_t n );
static inline double  gtod();

// Kernel that executes on the CUDA device
__global__ void multMatrixElement(uint32_t *a, uint32_t *b, uint32_t *c, int dim)
{
//	int row = threadIdx.x + BLOCKDIMX * blockIdx.y;	
//  int column = blockIdx.x;
	int row = threadIdx.y + blockIdx.y * BLOCKDIMY;	
	int column = threadIdx.x + blockIdx.x * BLOCKDIMX;
  	
 	if (row<dim && column<dim){
 		uint32_t row_mult_dim, j;
		uint32_t summe = (uint32_t) 0;
		
		row_mult_dim = row * dim;
		
        for ( j = 0; j < dim; j++ )
        {
            // C[i][j] += A[i][k] * B[k][j]
            summe += a[ row_mult_dim + j ] * b[ j * dim +  column];
        }
        c[ row_mult_dim + column ] = summe;
 	}
}

// main routine that executes on the host
int main(void)
{
	double t_start, t_end;	
	
	uint32_t *a_device, *a_host;  
	uint32_t *b_device, *b_host; 
	uint32_t *c_device, *c_host;  
	uint32_t *c_test;  

	const int dim = 1024;  // dimension of matrix
	const int N = dim*dim;  // Number of elements in arrays
	size_t size = N * sizeof(uint32_t);

	// Allocate array on host
	a_host = (uint32_t *)malloc(size);  
	b_host = (uint32_t *)malloc(size); 
	c_host = (uint32_t *)malloc(size); 
	c_test = (uint32_t *)malloc(size); 

	// Allocate array on device
	CudaSafeCall(hipMalloc((void **) &a_device, size)); CudaCheckError();   
	CudaSafeCall(hipMalloc((void **) &b_device, size)); CudaCheckError();   
	CudaSafeCall(hipMalloc((void **) &c_device, size)); CudaCheckError();   

	// Initialize host array and copy it to CUDA device
	a_host = random_mat( dim );
	b_host = random_mat( dim );
	c_host = zero_mat( dim );
	c_test = zero_mat( dim );

	CudaSafeCall(hipMemcpy(a_device, a_host, size, hipMemcpyHostToDevice)); CudaCheckError();
	CudaSafeCall(hipMemcpy(b_device, b_host, size, hipMemcpyHostToDevice)); CudaCheckError();
	CudaSafeCall(hipMemcpy(c_device, c_host, size, hipMemcpyHostToDevice)); CudaCheckError();

	// define grid and block sizes:
	dim3 griddim(GRIDDIMX, GRIDDIMY, GRIDDIMZ); 
	dim3 blockdim(BLOCKDIMX, BLOCKDIMY, BLOCKDIMZ); 
	
    printf("GridDim: x: %d y: %d z: %d \n",griddim.x,griddim.y,griddim.z);
    printf("BlockDim: x: %d y: %d z: %d \n",blockdim.x,blockdim.y,blockdim.z);

    t_start = gtod();
    
	// Do calculation on device:
	multMatrixElement <<< griddim, blockdim >>> (a_device, b_device, c_device, dim);
	CudaCheckError();
	
	hipDeviceSynchronize(); t_end = gtod();
    
	// Retrieve result from device and store it in host array
	CudaSafeCall(hipMemcpy(c_host, c_device, size, hipMemcpyDeviceToHost)); CudaCheckError();
	
    float gflops = ( ( double )2 * dim * dim * dim / 1000000000.0 ) / ( t_end - t_start );

    printf("Dim: %4d  runtime: %7.4fs  GFLOP/s: %0.2f\n", dim, t_end - t_start, gflops );
	
	uint32_t i_mult_dim, i_mult_dim_add_k, k_mult_dim, i, j, k;
		
    /* Begin matrix matrix multiply kernel */
    for ( i = 0; i < dim; i++ )
    {
		i_mult_dim = i * dim;
        for ( k = 0; k < dim; k++ )
        {
			i_mult_dim_add_k = i_mult_dim + k;
			k_mult_dim = k * dim;
	        for ( j = 0; j < dim; j++ )
	        {
                // C[i][j] += A[i][k] * B[k][j]
                c_test[ i_mult_dim + j ] += a_host[ i_mult_dim_add_k ] * b_host[ k_mult_dim + j ];
            }
        }
    }
    /* End matrix matrix multiply kernel */
		
	// Print results
	bool testOk = true;
	for (int i=0; i<N; i++){
		if(c_host[i] != c_test[i]) {
			printf("%d: %d, %d: %20d != %20d\n", i, (int) floor(i/dim), (i%dim), c_host[i], c_test[i]);
			testOk = false;
		}
	}
	
	if(testOk) printf("TEST PASSED\n"); else printf("TEST FAILED\n");


	// Cleanup
	free(a_host); CudaSafeCall(hipFree(a_device)); CudaCheckError();
	free(b_host); CudaSafeCall(hipFree(b_device)); CudaCheckError();
	free(c_host); CudaSafeCall(hipFree(c_device)); CudaCheckError();
	free(c_test);
}



/** @brief Get current timestamp in seconds.
 *
 *  @return         Returns current time stamp in seconds.
 */
static inline double gtod( )
{
    struct timeval act_time;
    gettimeofday( &act_time, NULL );

    return ( double )act_time.tv_sec + ( double )act_time.tv_usec / 1000000.0;
}

/** @brief Generate randomized matrix.
 *
 *  @param dim      Dimension for the generated matrix.
 *
 *  @return         Returns a pointer to the generated matrix on success, NULL
 *                  otherwise.
 */
static inline uint32_t* random_mat( uint32_t dim )
{
    uint32_t *matrix = ( uint32_t* )malloc( sizeof( uint32_t ) * dim * dim );
    if ( matrix == NULL )
    {
        return NULL;
    }

    srand( ( unsigned ) time( NULL ) );

    for ( uint32_t i = 0; i < dim * dim; ++i)
    {
        matrix[ i ] = ( uint32_t )rand();
    }

  return matrix;
}


/** @brief Generate zero matrix.
 *
 *  @param dim      Dimension for the generated matrix.
 *
 *  @return         Returns a pointer to the generated matrix on success, NULL
 *                  otherwise.
 */
static inline uint32_t* zero_mat( uint32_t dim )
{
    uint32_t* matrix = ( uint32_t* )malloc( sizeof( uint32_t ) * dim * dim );
    if ( matrix == NULL )
    {
        return NULL;
    }

    for ( uint32_t i = 0; i < dim * dim; ++i)
    {
        matrix[ i ] = ( uint32_t )0;
    }

  return matrix;
}

