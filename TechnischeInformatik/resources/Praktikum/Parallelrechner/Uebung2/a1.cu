//#include "stdafx.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "errorCheck.h"


// main routine that executes on the host
int main(void)
{
	int *a_host;  

	const int N = 1000000;  // Number of elements in arrays
	size_t size = N * sizeof(int);

	// Allocate arrays on host
	a_host = (int *)malloc(size);  
 

	// Initialize host arrays and copy it to CUDA device
	for (int i=0; i<N; i++){ 
		a_host[i] = (int)i; 
	}
  
}
