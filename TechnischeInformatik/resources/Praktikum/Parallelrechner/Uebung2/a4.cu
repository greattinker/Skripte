#include "hip/hip_runtime.h"
//#include "stdafx.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "errorCheck.h"

#define GRIDDIMX 1954
#define GRIDDIMY 1
#define GRIDDIMZ 1

#define BLOCKDIMX 512
#define BLOCKDIMY 1
#define BLOCKDIMZ 1


// Kernel that executes on the CUDA device
__global__ void addElems(int *a, int *b, int *c, int N)
{
  	int idx = (threadIdx.x + threadIdx.y * BLOCKDIMX + threadIdx.z * BLOCKDIMX * BLOCKDIMY) + (BLOCKDIMX*BLOCKDIMY*BLOCKDIMZ) * ( blockIdx.x + blockIdx.y * GRIDDIMX + blockIdx.z * GRIDDIMX * GRIDDIMY);
//	int idx = threadIdx.x + blockDim.x * blockIdx.x ;
  	if (idx<N) c[idx] = a[idx] + b[idx];
}

// main routine that executes on the host
int main(void)
{
	int *a_device, *a_host;  
	int *b_device, *b_host; 
	int *c_device, *c_host;  

	const int N = 1000000;  // Number of elements in arrays
	size_t size = N * sizeof(int);

	// Allocate arrays on host
	a_host = (int *)malloc(size);  
	b_host = (int *)malloc(size); 
	c_host = (int *)malloc(size); 

	// Allocate arrays on device
	CudaSafeCall(hipMalloc((void **) &a_device, size)); CudaCheckError();   
	CudaSafeCall(hipMalloc((void **) &b_device, size)); CudaCheckError();  
	CudaSafeCall(hipMalloc((void **) &c_device, size)); CudaCheckError();  

	// Initialize host arrays and copy it to CUDA device
	for (int i=0; i<N; i++){ 
		a_host[i] = (int)i; 
		b_host[i] = (int)i;
	}
  
	CudaSafeCall(hipMemcpy(a_device, a_host, size, hipMemcpyHostToDevice)); CudaCheckError();
	CudaSafeCall(hipMemcpy(b_device, b_host, size, hipMemcpyHostToDevice)); CudaCheckError();

	// Do calculation on device:
	dim3 griddim(GRIDDIMX, GRIDDIMY, GRIDDIMZ); 
	dim3 blockdim(BLOCKDIMX, BLOCKDIMY, BLOCKDIMZ); 
  
  	addElems <<< griddim, blockdim >>> (a_device, b_device, c_device, N);
  
  
  	// Retrieve result from device and store it in host array
 	 CudaSafeCall(hipMemcpy(c_host, c_device, size, hipMemcpyDeviceToHost));
	//CudaCheckError();

	// Print results
	for (int i=0; i<N; i++) if((a_host[i]+b_host[i]) != c_host[i])printf("%d %d %d %d\n", i, a_host[i], b_host[i], c_host[i]);


	// Cleanup
	free(a_host); CudaSafeCall(hipFree(a_device)); CudaCheckError();
	free(b_host); CudaSafeCall(hipFree(b_device)); CudaCheckError();
	free(c_host); CudaSafeCall(hipFree(c_device)); CudaCheckError();
}
